#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "raycast.h"

#include <cuda/Box3f.h>
#include <cuda/Event.h>
#include <cuda/MathUtils.h>
#include <cuda/ThreadMath.cuh>

#include "camera_math.cuh"
#include "tsdf.h"

using libcgt::cuda::contains;
using libcgt::cuda::Event;
using libcgt::cuda::math::floorToInt;
using libcgt::cuda::threadmath::threadSubscript2DGlobal;

__inline__ __device__ __host__
float2 half2()
{
    return make_float2(0.5f);
}

__inline__ __device__ __host__
float3 half3()
{
    return make_float3(0.5f);
}

__inline__ __device__ __host__
float2 one2()
{
    return make_float2(1.0f);
}

__inline__ __device__ __host__
float3 one3()
{
    return make_float3(1.0f);
}

// TODO: consider optimizing this by removing all boundary checks and
// adjusting the kernel.

// TODO: make this a method.
// TODO: easy way to enforce boundary conditions:
// clamp x to 0.5, width - 0.5,, etc.
// But that's not useful for SDFs! When you want to know when you're invalid.
// Assumes the grid_point is valid (coords are within the extents).
__inline__ __device__
float2 TrilinearSample(KernelArray3D<const TSDF> regular_grid,
  float3 grid_coords, float max_tsdf_value) {
  // For trilinear interpolation, the valid range is between [0.5, size - 0.5].
  libcgt::cuda::Box3f valid_box(half3(),
    make_float3(regular_grid.size()) - one3());
  if (!valid_box.contains(grid_coords)) {
    return{ 0.0f, 0.0f };
  }

  //
  float3 integer_grid_coords = grid_coords - half3();
  int3 p_000 = floorToInt(integer_grid_coords);
  float3 t = fracf(integer_grid_coords);
  int3 p_100 = { p_000.x + 1, p_000.y,     p_000.z     };
  int3 p_010 = { p_000.x    , p_000.y + 1, p_000.z     };
  int3 p_110 = { p_000.x + 1, p_000.y + 1, p_000.z     };
  int3 p_001 = { p_000.x    , p_000.y    , p_000.z + 1 };
  int3 p_101 = { p_000.x + 1, p_000.y    , p_000.z + 1 };
  int3 p_011 = { p_000.x    , p_000.y + 1, p_000.z + 1 };
  int3 p_111 = { p_000.x + 1, p_000.y + 1, p_000.z + 1 };

  TSDF v_000 = regular_grid[p_000];
  TSDF v_100 = regular_grid[p_100];
  TSDF v_010 = regular_grid[p_010];
  TSDF v_110 = regular_grid[p_110];
  TSDF v_001 = regular_grid[p_001];
  TSDF v_101 = regular_grid[p_101];
  TSDF v_011 = regular_grid[p_011];
  TSDF v_111 = regular_grid[p_111];

  // TODO(jiawen): can save a branch by multiplying by weight, or maybe storing
  // pre-multiplied.
  if (v_000.Weight() == 0 || v_100.Weight() == 0 ||
    v_010.Weight() == 0 || v_110.Weight() == 0 ||
    v_001.Weight() == 0 || v_101.Weight() == 0 ||
    v_011.Weight() == 0 || v_111.Weight() == 0) {
    return{ 0.0f, 0.0f };
  }

  // Trilerp, ignoring weights.
  // TODO(jiawen): weighted sdf?
  float d_000 = v_000.Distance(max_tsdf_value);
  float d_100 = v_100.Distance(max_tsdf_value);
  float d_010 = v_010.Distance(max_tsdf_value);
  float d_110 = v_110.Distance(max_tsdf_value);
  float d_001 = v_001.Distance(max_tsdf_value);
  float d_101 = v_101.Distance(max_tsdf_value);
  float d_011 = v_011.Distance(max_tsdf_value);
  float d_111 = v_111.Distance(max_tsdf_value);

  // Lerp in x.
  float d_l00 = lerp(d_000, d_100, t.x);
  float d_l10 = lerp(d_010, d_110, t.x);
  float d_l01 = lerp(d_000, d_101, t.x);
  float d_l11 = lerp(d_010, d_111, t.x);

  // Lerp in y.
  float d_ll0 = lerp(d_l00, d_l10, t.y);
  float d_ll1 = lerp(d_l01, d_l11, t.y);

  // Lerp in z.
  return { lerp(d_ll0, d_ll1, t.z), 1.0f };
}

// TODO(jiawen): optimized version without checks?
__inline__ __device__
float4 TrilinearSampleNormal(KernelArray3D<const TSDF> regular_grid,
  float3 grid_coords, float max_tsdf_value) {
  float3 dx3 = { 1, 0, 0 };
  float3 dy3 = { 0, 1, 0 };
  float3 dz3 = { 0, 0, 1 };

  float2 d_000 = TrilinearSample(regular_grid, grid_coords, max_tsdf_value);
  float2 d_100 = TrilinearSample(regular_grid, grid_coords + dx3, max_tsdf_value);
  float2 d_010 = TrilinearSample(regular_grid, grid_coords + dy3, max_tsdf_value);
  float2 d_001 = TrilinearSample(regular_grid, grid_coords + dz3, max_tsdf_value);
  bool valid = d_000.y != 0 && d_100.y != 0 && d_010.y != 0 && d_001.y != 0;

  float4 normal_out = {};

  float3 normal = {
    d_100.x - d_000.x,
    d_010.x - d_000.x,
    d_001.x - d_000.x,
  };
  float len = length(normal);
  if (valid && len > 0) {
    normal_out = make_float4(normal / len, 1.0f);
  }

  return normal_out;
}

#define kTEpsilon 2.0f
#define kTStepSize 1.0f

__global__
void RaycastKernel(KernelArray3D<const TSDF> regular_grid,
  float4x4 grid_from_world,
  float4x4 world_from_grid,
  float max_tsdf_value,
  float4 flpp,
  float4x4 world_from_camera,
  float3 eye_world,
  KernelArray2D<float4> world_points_out,
  KernelArray2D<float4> world_normals_out) {
  // TODO(jiawen): simplify this logic with a "bool valid" flag.
  float4 world_point = {};
  float4 world_normal = {};

  // Cast a ray for each pixel.
  int2 xy = threadSubscript2DGlobal();
  if (!contains(world_points_out.size(), xy)) {
    world_points_out[xy] = world_point;
    world_normals_out[xy] = world_normal;
    return;
  }

  float3 dir_grid = normalize(transformVector(grid_from_world,
    transformVector(world_from_camera, CameraDirectionFromPixel(xy, flpp))));

  // TODO(jiawen): make this a method, or pass it in directly
  float3 eye_grid = transformPoint(grid_from_world, eye_world);

  // Pick a starting point: intersect the ray with the grid bounding box.
  float t_near;
  float t_far;
  // TODO(jiawen): intersect with a grid that's 1 voxel smaller.
  libcgt::cuda::Box3f bbox_grid(regular_grid.size());
  bool intersected = libcgt::cuda::intersectLine(eye_grid, dir_grid,
    bbox_grid, t_near, t_far);

  if (!intersected) {
    world_points_out[xy] = world_point;
    world_normals_out[xy] = world_normal;
    return;
  }

  // If the near starting point is behind the eye, clamp it to the eye.
  // If it's in front of the eye, then start there.
  // But we don't want to start directly on a face, so add epsilon to it.
  float t_start = fmaxf(0, t_near) + kTEpsilon;

  // Likewise, the end point should not be on a face.
  float t_end = fmaxf(0, t_far) - kTEpsilon;

  int num_iterations = floorToInt((t_end - t_start) / kTStepSize);

  // Iterate until we exit, or found a surface.
  bool found_surface = false;

  float prev_t;
  float3 prev_coords_grid = {};
  float2 prev_sdf = {};

  float curr_t = t_near;
  float3 curr_coords_grid = eye_grid + curr_t * dir_grid;
  float2 curr_sdf =
    TrilinearSample(regular_grid, curr_coords_grid, max_tsdf_value);

  for (int i = 1; i < num_iterations; ++i) {
    prev_t = curr_t;
    prev_coords_grid = curr_coords_grid;
    prev_sdf = curr_sdf;

    curr_t = prev_t + kTStepSize;
    curr_coords_grid = eye_grid + curr_t * dir_grid;
    curr_sdf = TrilinearSample(regular_grid, curr_coords_grid, max_tsdf_value);

    // Both samples are valid, and it's a positive to negative zero crossing.
    if (prev_sdf.y > 0 && curr_sdf.y > 0 &&
      prev_sdf.x > 0 && curr_sdf.x < 0) {
      found_surface = true;
      break;
    }
  }

  if (found_surface) {
    // How far should I interpolate between the SDF values?
    float alpha = prev_sdf.x / (prev_sdf.x - curr_sdf.x);

    // Use it to lerp t itself to get a better estimate of the zero crossing.
    float t_at_surface = lerp(prev_t, curr_t, alpha);

    float3 surface_point_grid = eye_grid + t_at_surface * dir_grid;

    // Convert to world space.
    // TODO(jiawen): make this a method
    world_point = make_float4(
      transformPoint(world_from_grid, surface_point_grid), 1.0f);
    float4 grid_normal = TrilinearSampleNormal(regular_grid,
      surface_point_grid, max_tsdf_value);
    if (grid_normal.w > 0) {
      // TODO(jiawen): the gradient is weird and only needs the rotation part
      // of the world_from_grid transformation. It's because we store world
      // distances in the grid.
      world_normal = make_float4(
        normalize(transformVector(world_from_grid, make_float3(grid_normal))),
        1.0f);
    }
  }

  world_points_out[xy] = world_point;
  world_normals_out[xy] = world_normal;
}
