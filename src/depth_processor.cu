#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "depth_processor.h"

#include <cuda/Event.h>
#include <cuda/MathUtils.h>
#include <cuda/Rect2i.h>
#include <cuda/ThreadMath.cuh>
#include <cuda/VecmathConversions.h>

#include "camera_math.cuh"

using libcgt::cuda::Event;
using libcgt::cuda::threadmath::threadSubscript2DGlobal;
using libcgt::cuda::contains;
using libcgt::cuda::inset;
using libcgt::cuda::Rect2i;
using libcgt::cuda::math::numBins2D;

__global__
void SmoothDepthMapKernel(KernelArray2D<const float> input,
  float2 depth_min_max,
  int kernel_radius,
  float delta_z_squared_threshold,
  KernelArray2D<float> smoothed) {
  int2 xy = threadSubscript2DGlobal();
  Rect2i valid_rect = inset(Rect2i(input.size()),
    { kernel_radius, kernel_radius });
  float z = input[xy];
  float smoothed_z = 0.0f;
  if (contains(valid_rect, xy) &&
    z >= depth_min_max.x && z <= depth_min_max.y) {

    float sum = 0.0f;
    float sum_weights = 0.0f;

    for (int dy = -kernel_radius; dy <= kernel_radius; ++dy) {
      for (int dx = -kernel_radius; dx <= kernel_radius; ++dx) {
        float z2 = input[xy + int2{dx, dy}];
        float delta_z = z2 - z;
        float delta_z_squared = delta_z * delta_z;
        if (z2 != 0 && delta_z_squared < delta_z_squared_threshold) {
          float dr2 = dx * dx + dy * dy;
          float dr = sqrt(dr2);
          // TODO(jiawen): Hacky bilateral filter without exp().
          float spatial_weight = 1.0f / (1.0f + dr);
          float range_weight = delta_z_squared_threshold - delta_z_squared;
          float weight = spatial_weight * range_weight;
          sum += weight * z2;
          sum_weights += weight;
        }
      }
    }

    if (sum_weights > 0.0f) {
      smoothed_z = sum / sum_weights;
    }
  }
  smoothed[xy] = smoothed_z;
}

__global__
void UndistortKernel(hipTextureObject_t raw_depth,
  hipTextureObject_t undistort_map,
  KernelArray2D<float> undistorted) {
  int2 xy = threadSubscript2DGlobal();
  if (contains(Rect2i(undistorted.size()), xy)) {
    // TODO: make a function to go from xy to [0,1]

    float u = xy.x + 0.5f;
    float v = xy.y + 0.5f;

    // Fetch from undistort_map[uv] to get xy2.
    float2 xy2 = tex2D<float2>(undistort_map, u, v);

    undistorted[xy] = tex2D<float>(raw_depth, xy2.x, xy2.y);
  }
}

__global__
void EstimateNormalsKernel(KernelArray2D<const float> depth_map,
  float4 flpp, float2 depth_min_max,
  KernelArray2D<float4> normals) {
  int2 xy = threadSubscript2DGlobal();
  float4 normal = {};

  if (xy.x < depth_map.width() - 1 && xy.y < depth_map.height() - 1) {
    float depth0 = depth_map[xy];
    int2 xy1{ xy.x + 1, xy.y };
    int2 xy2{ xy.x, xy.y + 1 };
    float depth1 = depth_map[xy1];
    float depth2 = depth_map[xy2];

    if (depth0 >= depth_min_max.x && depth0 <= depth_min_max.y &&
      depth1 >= depth_min_max.x && depth1 <= depth_min_max.y &&
      depth2 >= depth_min_max.x && depth2 <= depth_min_max.y) {

      // TODO: can optimize this by not using CameraFromPixel and directly
      // scaling x and y by z.
      float3 p0 = CameraFromPixel(xy, depth0, flpp);
      float3 p1 = CameraFromPixel(xy1, depth1, flpp);
      float3 p2 = CameraFromPixel(xy2, depth2, flpp);

      float3 dx = p1 - p0;
      float3 dy = p2 - p0;
      float3 n = cross(dx, dy);
      float lenSquared = lengthSquared(n);
      if (lenSquared > 0.0f) {
        normal = make_float4(n / sqrt(lenSquared), 1.0f);
      }
    }
  }

  normals[xy] = normal;
}

DepthProcessor::DepthProcessor(const Intrinsics& depth_intrinsics,
  const Range1f& depth_range) :
  depth_intrinsics_flpp_{ depth_intrinsics.focalLength,
    depth_intrinsics.principalPoint },
  depth_range_(depth_range) {

}

void DepthProcessor::Undistort(DeviceArray2D<float>& raw_depth,
  DeviceArray2D<float2>& undistort_map,
  DeviceArray2D<float>& undistorted_depth) {
  // Bind raw_depth and undistort_map to texture objects.

  hipResourceDesc raw_depth_res_desc = raw_depth.resourceDesc();
  hipResourceDesc undistort_map_res_desc = undistort_map.resourceDesc();

  hipTextureDesc point_normalized_tex_desc = {};
  point_normalized_tex_desc.addressMode[0] = hipAddressModeClamp;
  point_normalized_tex_desc.addressMode[1] = hipAddressModeClamp;
  point_normalized_tex_desc.filterMode = hipFilterModePoint;
  point_normalized_tex_desc.readMode = hipReadModeElementType;
  point_normalized_tex_desc.normalizedCoords = true;

  hipTextureDesc point_unnormalized_tex_desc = {};
  point_unnormalized_tex_desc.addressMode[0] = hipAddressModeClamp;
  point_unnormalized_tex_desc.addressMode[1] = hipAddressModeClamp;
  point_unnormalized_tex_desc.filterMode = hipFilterModePoint;
  point_unnormalized_tex_desc.readMode = hipReadModeElementType;
  point_unnormalized_tex_desc.normalizedCoords = false;

  hipError_t err;

  hipTextureObject_t raw_depth_tex_obj;
  err = hipCreateTextureObject(&raw_depth_tex_obj, &raw_depth_res_desc,
      &point_normalized_tex_desc, nullptr);

  hipTextureObject_t undistort_map_tex_obj;
  err = hipCreateTextureObject(&undistort_map_tex_obj, &undistort_map_res_desc,
      &point_unnormalized_tex_desc, nullptr);

  dim3 block(16, 16);
  dim3 grid = numBins2D(make_int2(raw_depth.size()), block);

  Event e;
  e.recordStart();
  UndistortKernel<<<grid, block>>>(
    raw_depth_tex_obj,
    undistort_map_tex_obj,
    undistorted_depth.writeView());
  float dtMS = e.recordStopSyncAndGetMillisecondsElapsed();
  printf("DepthProcessor::Undistort took %f ms\n", dtMS);

  // TODO: don't destroy the texture every time.
  hipDestroyTextureObject(undistort_map_tex_obj);
  hipDestroyTextureObject(raw_depth_tex_obj);
}

void DepthProcessor::Smooth(DeviceArray2D<float>& raw_depth,
  DeviceArray2D<float>& smoothed_depth) {

  dim3 block(16, 16);
  dim3 grid = numBins2D(make_int2(raw_depth.size()), block);

  Event e;
  e.recordStart();
  SmoothDepthMapKernel<<<grid, block>>>(
    raw_depth.readView(),
    make_float2(depth_range_.leftRight()),
    kernel_radius_,
    delta_z_squared_threshold_,
    smoothed_depth.writeView());
  float dtMS = e.recordStopSyncAndGetMillisecondsElapsed();
  printf("DepthProcessor::Smooth took %f ms\n", dtMS);
}

void DepthProcessor::EstimateNormals(DeviceArray2D<float>& smoothed_depth,
  DeviceArray2D<float4>& normals) {
  dim3 block(16, 16);
  dim3 grid = numBins2D(make_int2(smoothed_depth.size()), block);

  Event e;
  e.recordStart();
  EstimateNormalsKernel<<<grid, block>>>(
    smoothed_depth.readView(),
    make_float4(depth_intrinsics_flpp_),
    make_float2(depth_range_.leftRight()),
    normals.writeView());
  float dtMS = e.recordStopSyncAndGetMillisecondsElapsed();
  printf("DepthProcessor::EstimateNormals took %f ms\n", dtMS);
}
