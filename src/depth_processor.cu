#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "depth_processor.h"

#include <cuda/Event.h>
#include <CUDA/MathUtils.h>
#include <CUDA/Rect2i.h>
#include <CUDA/ThreadMath.cuh>
#include <cuda/VecmathConversions.h>

#include "camera_math.cuh"

using libcgt::cuda::Event;
using libcgt::cuda::threadmath::threadSubscript2DGlobal;
using libcgt::cuda::contains;
using libcgt::cuda::inset;
using libcgt::cuda::Rect2i;
using libcgt::cuda::math::numBins2D;

__global__
void SmoothDepthMapKernel(KernelArray2D<const float> input,
  float2 depth_min_max,
  int kernel_radius,
  float delta_z_squared_threshold,
  KernelArray2D<float> smoothed) {
  int2 xy = threadSubscript2DGlobal();
  Rect2i valid_rect = inset(Rect2i(input.size()),
    { kernel_radius, kernel_radius });
  float z = input[xy];
  float smoothed_z = 0.0f;
  if (contains(valid_rect, xy) &&
    z >= depth_min_max.x && z <= depth_min_max.y) {

    float sum = 0.0f;
    float sum_weights = 0.0f;

    for (int dy = -kernel_radius; dy <= kernel_radius; ++dy) {
      for (int dx = -kernel_radius; dx <= kernel_radius; ++dx) {
        float z2 = input[xy + int2{dx, dy}];
        float delta_z = z2 - z;
        float delta_z_squared = delta_z * delta_z;
        if (z2 != 0 && delta_z_squared < delta_z_squared_threshold) {
          float dr2 = dx * dx + dy * dy;
          float dr = sqrt(dr2);
          // TODO(jiawen): Hacky bilateral filter without exp().
          float spatial_weight = 1.0f / (1.0f + dr);
          float range_weight = delta_z_squared_threshold - delta_z_squared;
          float weight = spatial_weight * range_weight;
          sum += weight * z2;
          sum_weights += weight;
        }
      }
    }

    if (sum_weights > 0.0f) {
      smoothed_z = sum / sum_weights;
    }
  }
  smoothed[xy] = smoothed_z;
}

__global__
void EstimateNormalsKernel(KernelArray2D<const float> depth_map,
  float4 flpp, float2 depth_min_max,
  KernelArray2D<float4> normals) {
  int2 xy = threadSubscript2DGlobal();
  float4 normal = {};

  if (xy.x < depth_map.width() - 1 && xy.y < depth_map.height() - 1) {
    float depth0 = depth_map[xy];
    int2 xy1{ xy.x + 1, xy.y };
    int2 xy2{ xy.x, xy.y + 1 };
    float depth1 = depth_map[xy1];
    float depth2 = depth_map[xy2];

    if (depth0 >= depth_min_max.x && depth0 <= depth_min_max.y &&
      depth1 >= depth_min_max.x && depth1 <= depth_min_max.y &&
      depth2 >= depth_min_max.x && depth2 <= depth_min_max.y) {

      // TODO(jiawen): can optimize this by not using CameraFromPixel and just
      // scaling x and y by z.
      float3 p0 = CameraFromPixel(xy, depth0, flpp);
      float3 p1 = CameraFromPixel(xy1, depth1, flpp);
      float3 p2 = CameraFromPixel(xy2, depth2, flpp);

      float3 dx = p1 - p0;
      float3 dy = p2 - p0;
      float3 n = cross(dx, dy);
      float lenSquared = lengthSquared(n);
      if (lenSquared > 0.0f) {
        normal = make_float4(n / sqrt(lenSquared), 1.0f);
      }
    }
  }

  normals[xy] = normal;
}

DepthProcessor::DepthProcessor(const Intrinsics& depth_intrinsics,
  const Range1f& depth_range) :
  depth_intrinsics_flpp_{ depth_intrinsics.focalLength,
    depth_intrinsics.principalPoint },
  depth_range_(depth_range) {

}

void DepthProcessor::SmoothDepth(DeviceArray2D<float>& raw_depth,
  DeviceArray2D<float>& smoothed_depth) {

  dim3 block(16, 16);
  dim3 grid = numBins2D(make_int2(raw_depth.size()), block);

  Event e;
  SmoothDepthMapKernel<<<grid, block>>>(
    raw_depth.readView(),
    make_float2(depth_range_.leftRight()),
    kernel_radius_,
    delta_z_squared_threshold_,
    smoothed_depth.writeView());
  float dtMS = e.recordStopSyncAndGetMillisecondsElapsed();
  printf("SmoothDepth took %f ms\n", dtMS);
}

void DepthProcessor::EstimateNormals(DeviceArray2D<float>& smoothed_depth,
  DeviceArray2D<float4>& normals) {
  dim3 block(16, 16);
  dim3 grid = numBins2D(make_int2(smoothed_depth.size()), block);

  Event e;
  EstimateNormalsKernel<<<grid, block>>>(
    smoothed_depth.readView(),
    make_float4(depth_intrinsics_flpp_),
    make_float2(depth_range_.leftRight()),
    normals.writeView());
  float dtMS = e.recordStopSyncAndGetMillisecondsElapsed();
  printf("EstimateNormals took %f ms\n", dtMS);
}
