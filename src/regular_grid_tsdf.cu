#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "regular_grid_tsdf.h"

#include <cassert>

#include <gflags/gflags.h>

#include "libcgt/cuda/Event.h"
#include "libcgt/cuda/MathUtils.h"
#include "libcgt/cuda/VecmathConversions.h"

#include "fuse.h"
#include "marching_cubes.h"
#include "raycast.h"

using libcgt::core::vecmath::SimilarityTransform;
using libcgt::core::vecmath::inverse;
using libcgt::cuda::Event;

DECLARE_bool(collect_perf);

// VoxelSize() = world_from_grid_.scale.
RegularGridTSDF::RegularGridTSDF(const Vector3i& resolution,
  const SimilarityTransform& world_from_grid) :
  RegularGridTSDF(resolution, world_from_grid, 4 * world_from_grid.scale) {
}

RegularGridTSDF::RegularGridTSDF(const Vector3i& resolution,
  const SimilarityTransform& world_from_grid, float max_tsdf_value) :
  device_grid_(resolution),
  bounding_box_(resolution),
  world_from_grid_(world_from_grid),
  grid_from_world_(inverse(world_from_grid)),
  max_tsdf_value_(max_tsdf_value) {
  assert(VoxelSize() > 0);
  assert(max_tsdf_value > 0);

  Reset();
}

void RegularGridTSDF::Reset() {
  TSDF empty(0, 0, max_tsdf_value_);
  device_grid_.fill(empty);
}

const SimilarityTransform& RegularGridTSDF::GridFromWorld() const {
  return grid_from_world_;
}

const SimilarityTransform& RegularGridTSDF::WorldFromGrid() const {
  return world_from_grid_;
}

Box3f RegularGridTSDF::BoundingBox() const {
  return bounding_box_;
}

Vector3i RegularGridTSDF::Resolution() const {
  return device_grid_.size();
}

float RegularGridTSDF::VoxelSize() const {
  return world_from_grid_.scale;
}

Vector3f RegularGridTSDF::SideLengths() const {
  return VoxelSize() * Resolution();
}

void RegularGridTSDF::Fuse(const Vector4f& depth_camera_flpp,
  const Range1f& depth_range,
  const Matrix4f& camera_from_world,
  const DeviceArray2D<float>& depth_data) {

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { device_grid_.width(), device_grid_.height() },
    block_dim
  );

  // TODO: move these into class or use Performance Collector class.
  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  FuseKernel<<<grid_dim, block_dim>>>(
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    make_float4(depth_camera_flpp),
    make_float2(depth_range.left(), depth_range.right()),
    make_float4x4(camera_from_world),
    depth_data.readView(),
    device_grid_.writeView());

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("Fuse() took: %f ms\n", msElapsed);

    printf("%d average: %f\n", nIterationsTotal, msTotal / nIterationsTotal);
    printf("3x average: %f\n", 3.0f * msTotal / nIterationsTotal);
  }
}

void RegularGridTSDF::FuseMultiple(
  const std::vector<CalibratedPosedDepthCamera>& depth_cameras,
  const std::vector<DeviceArray2D<float>>& depth_maps) {
  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { device_grid_.width(), device_grid_.height() },
    block_dim
  );

  // TODO: move these into class or use Performance Collector class.
  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  FuseMultipleKernel<<<grid_dim, block_dim>>>(
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    depth_cameras[0],
    depth_cameras[1],
    depth_cameras[2],
    depth_maps[0].readView(),
    depth_maps[1].readView(),
    depth_maps[2].readView(),
    device_grid_.writeView());

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("FuseMultiple() took: %f ms, %d-run average: %f\n",
      msElapsed, nIterationsTotal, msTotal / nIterationsTotal);
  }
}

void RegularGridTSDF::AdaptiveRaycast(const Vector4f& depth_camera_flpp,
  const Matrix4f& world_from_camera,
  DeviceArray2D<float4>& world_points_out,
  DeviceArray2D<float4>& world_normals_out) {
  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { world_points_out.width(), world_points_out.height() },
    block_dim
  );

  Vector4f eye = world_from_camera * Vector4f(0, 0, 0, 1);
  float voxels_per_meter = 1.0f / VoxelSize();

  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  AdaptiveRaycastKernel<<<grid_dim, block_dim>>>(
    device_grid_.readView(),
    make_float4x4(grid_from_world_.asMatrix()),
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    voxels_per_meter,
    make_float4(depth_camera_flpp),
    make_float4x4(world_from_camera),
    make_float3(eye.xyz),
    world_points_out.writeView(),
    world_normals_out.writeView()
  );

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("AdaptiveRaycastKernel took: %f ms\n", msElapsed);

    printf("%d run average: %f ms\n",
      nIterationsTotal, msTotal / nIterationsTotal);
    printf("resolution: %d x %d\n", world_points_out.width(),
      world_points_out.height());
  }
}

void RegularGridTSDF::Raycast(const Vector4f& depth_camera_flpp,
  const Matrix4f& world_from_camera,
  DeviceArray2D<float4>& world_points_out,
  DeviceArray2D<float4>& world_normals_out) {

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { world_points_out.width(), world_points_out.height() },
    block_dim
  );

  Vector4f eye = world_from_camera * Vector4f(0, 0, 0, 1);

  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  RaycastKernel<<<grid_dim, block_dim>>>(
    device_grid_.readView(),
    make_float4x4(grid_from_world_.asMatrix()),
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    make_float4(depth_camera_flpp),
    make_float4x4(world_from_camera),
    make_float3(eye.xyz),
    world_points_out.writeView(),
    world_normals_out.writeView()
  );

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("RaycastKernel took: %f ms\n", msElapsed);

    printf("%d run average: %f ms\n",
      nIterationsTotal, msTotal / nIterationsTotal);
    printf("resolution: %d x %d\n", world_points_out.width(),
      world_points_out.height());
  }
}

TriangleMesh RegularGridTSDF::Triangulate() const {
  Array3D<TSDF> host_grid(Resolution());
  copy(device_grid_, host_grid.writeView());

  std::vector<Vector3f> positions;
  std::vector<Vector3f> normals;
  MarchingCubes(host_grid, max_tsdf_value_, world_from_grid_,
    positions, normals);

  return ConstructMarchingCubesMesh(positions, normals);
}
