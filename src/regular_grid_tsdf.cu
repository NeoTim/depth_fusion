#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "regular_grid_tsdf.h"

#include <cassert>

#include <gflags/gflags.h>

#include "libcgt/core/common/ArrayUtils.h"
#include "libcgt/core/io/BinaryFileInputStream.h"
#include "libcgt/core/io/BinaryFileOutputStream.h"
#include "libcgt/cuda/Event.h"
#include "libcgt/cuda/MathUtils.h"
#include "libcgt/cuda/VecmathConversions.h"

#include "fuse.h"
#include "marching_cubes.h"
#include "raycast.h"

using libcgt::core::arrayutils::flatten;
using libcgt::core::vecmath::SimilarityTransform;
using libcgt::core::vecmath::inverse;
using libcgt::cuda::Event;

DECLARE_bool(collect_perf);

// VoxelSize() = world_from_grid_.scale.
RegularGridTSDF::RegularGridTSDF(const Vector3i& resolution,
  const SimilarityTransform& world_from_grid) :
  RegularGridTSDF(resolution, world_from_grid, 4 * world_from_grid.scale) {
}

RegularGridTSDF::RegularGridTSDF(const Vector3i& resolution,
  const SimilarityTransform& world_from_grid, float max_tsdf_value) :
  device_grid_(resolution),
  world_from_grid_(world_from_grid),
  grid_from_world_(inverse(world_from_grid)),
  max_tsdf_value_(max_tsdf_value) {
  assert(VoxelSize() > 0);
  assert(max_tsdf_value > 0);

  Reset();
}

void RegularGridTSDF::Reset() {
  TSDF empty(0, 0, max_tsdf_value_);
  device_grid_.fill(empty);
}

const SimilarityTransform& RegularGridTSDF::GridFromWorld() const {
  return grid_from_world_;
}

const SimilarityTransform& RegularGridTSDF::WorldFromGrid() const {
  return world_from_grid_;
}

Box3f RegularGridTSDF::BoundingBox() const {
  return Box3f(device_grid_.size());
}

Vector3i RegularGridTSDF::Resolution() const {
  return device_grid_.size();
}

float RegularGridTSDF::VoxelSize() const {
  return world_from_grid_.scale;
}

Vector3f RegularGridTSDF::SideLengths() const {
  return VoxelSize() * Resolution();
}

void RegularGridTSDF::Fuse(const Vector4f& depth_camera_flpp,
  const Range1f& depth_range,
  const Matrix4f& camera_from_world,
  const DeviceArray2D<float>& depth_data) {

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { device_grid_.width(), device_grid_.height() },
    block_dim
  );

  // TODO: move these into class or use Performance Collector class.
  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  FuseKernel<<<grid_dim, block_dim>>>(
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    make_float4(depth_camera_flpp),
    make_float2(depth_range.left(), depth_range.right()),
    make_float4x4(camera_from_world),
    depth_data.readView(),
    device_grid_.writeView());

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("Fuse() took: %f ms\n", msElapsed);

    printf("%d average: %f\n", nIterationsTotal, msTotal / nIterationsTotal);
    printf("3x average: %f\n", 3.0f * msTotal / nIterationsTotal);
  }
}

void RegularGridTSDF::FuseMultiple(
  const std::vector<CalibratedPosedDepthCamera>& depth_cameras,
  const std::vector<DeviceArray2D<float>>& depth_maps) {
  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { device_grid_.width(), device_grid_.height() },
    block_dim
  );

  // TODO: move these into class or use Performance Collector class.
  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  FuseMultipleKernel<<<grid_dim, block_dim>>>(
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    depth_cameras[0],
    depth_cameras[1],
    depth_cameras[2],
    depth_maps[0].readView(),
    depth_maps[1].readView(),
    depth_maps[2].readView(),
    device_grid_.writeView());

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("FuseMultiple() took: %f ms, %d-run average: %f\n",
      msElapsed, nIterationsTotal, msTotal / nIterationsTotal);
  }
}

void RegularGridTSDF::AdaptiveRaycast(const Vector4f& depth_camera_flpp,
  const Matrix4f& world_from_camera,
  DeviceArray2D<float4>& world_points_out,
  DeviceArray2D<float4>& world_normals_out) {
  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { world_points_out.width(), world_points_out.height() },
    block_dim
  );

  Vector4f eye = world_from_camera * Vector4f(0, 0, 0, 1);
  float voxels_per_meter = 1.0f / VoxelSize();

  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  AdaptiveRaycastKernel<<<grid_dim, block_dim>>>(
    device_grid_.readView(),
    make_float4x4(grid_from_world_.asMatrix()),
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    voxels_per_meter,
    make_float4(depth_camera_flpp),
    make_float4x4(world_from_camera),
    make_float3(eye.xyz),
    world_points_out.writeView(),
    world_normals_out.writeView()
  );

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("AdaptiveRaycastKernel took: %f ms\n", msElapsed);

    printf("%d run average: %f ms\n",
      nIterationsTotal, msTotal / nIterationsTotal);
    printf("resolution: %d x %d\n", world_points_out.width(),
      world_points_out.height());
  }
}

void RegularGridTSDF::Raycast(const Vector4f& depth_camera_flpp,
  const Matrix4f& world_from_camera,
  DeviceArray2D<float4>& world_points_out,
  DeviceArray2D<float4>& world_normals_out) {

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { world_points_out.width(), world_points_out.height() },
    block_dim
  );

  Vector4f eye = world_from_camera * Vector4f(0, 0, 0, 1);

  static float msTotal = 0.0f;
  static int nIterationsTotal = 0;
  Event e;

  if (FLAGS_collect_perf) {
    e.recordStart();
  }

  RaycastKernel<<<grid_dim, block_dim>>>(
    device_grid_.readView(),
    make_float4x4(grid_from_world_.asMatrix()),
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    make_float4(depth_camera_flpp),
    make_float4x4(world_from_camera),
    make_float3(eye.xyz),
    world_points_out.writeView(),
    world_normals_out.writeView()
  );

  if (FLAGS_collect_perf) {
    float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

    msTotal += msElapsed;
    ++nIterationsTotal;

    printf("RaycastKernel took: %f ms\n", msElapsed);

    printf("%d run average: %f ms\n",
      nIterationsTotal, msTotal / nIterationsTotal);
    printf("resolution: %d x %d\n", world_points_out.width(),
      world_points_out.height());
  }
}

TriangleMesh RegularGridTSDF::Triangulate() const {
  Array3D<TSDF> host_grid(Resolution());
  copy(device_grid_, host_grid.writeView());

  std::vector<Vector3f> positions;
  std::vector<Vector3f> normals;
  MarchingCubes(host_grid, max_tsdf_value_, world_from_grid_,
    positions, normals);

  return ConstructMarchingCubesMesh(positions, normals);
}

bool RegularGridTSDF::Load(const std::string& filename) {
  // TODO: validate input at each step..
  BinaryFileInputStream in(filename);

  uint8_t header;
  in.read(header);
  in.read(header);
  in.read(header);
  in.read(header);
  in.read(header);
  in.read(header);

  int32_t version;
  in.read(version);

  Vector3i resolution;
  in.read(resolution);

  Matrix4f world_from_grid_matrix;
  in.read(world_from_grid_matrix);

  float max_tsdf_value;
  in.read(max_tsdf_value);

  Array3D<TSDF> data(resolution);
  in.readArray(flatten(data.writeView()));

  world_from_grid_ = SimilarityTransform::fromMatrix(world_from_grid_matrix);
  grid_from_world_ = inverse(world_from_grid_);

  copy(data.readView(), device_grid_);

  max_tsdf_value_ = max_tsdf_value;

  return true;
}

bool RegularGridTSDF::Save(const std::string& filename) const {
  // TODO: check ok after each write.
  BinaryFileOutputStream out(filename);

  // TODO: check that the stream is valid.

  // Write magic header: 'tsdf3d1'.
  out.write('t');
  out.write('s');
  out.write('d');
  out.write('f');
  out.write('3');
  out.write('d');
  out.write<int32_t>(1);

  // Write resolution: x, y, z.
  out.write(device_grid_.size());

  // Write world from grid transformation as a 4x4 float32 matrix,
  // stored column major.
  out.write(world_from_grid_.asMatrix());

  // Write max tsdf value.
  out.write(max_tsdf_value_);

  // Write data.
  Array3D<TSDF> data(device_grid_.size());
  copy(device_grid_, data.writeView());
  out.writeArray(flatten(data.readView()));

  return out.close();
}
