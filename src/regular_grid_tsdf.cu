#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "regular_grid_tsdf.h"

#include <cassert>

#include <cuda/Event.h>
#include <cuda/MathUtils.h>
#include <cuda/VecmathConversions.h>

#include "fuse.h"
#include "marching_cubes.h"
#include "raycast.h"

using libcgt::core::vecmath::SimilarityTransform;
using libcgt::core::vecmath::inverse;
using libcgt::cuda::Event;

// VoxelSize() = world_from_grid_.scale.
RegularGridTSDF::RegularGridTSDF(const Vector3i& resolution,
  const SimilarityTransform& world_from_grid) :
  RegularGridTSDF(resolution, world_from_grid, 4 * world_from_grid.scale) {
}

RegularGridTSDF::RegularGridTSDF(const Vector3i& resolution,
  const SimilarityTransform& world_from_grid, float max_tsdf_value) :
  device_grid_(resolution),
  bounding_box_(resolution),
  world_from_grid_(world_from_grid),
  grid_from_world_(inverse(world_from_grid)),
  max_tsdf_value_(max_tsdf_value) {
  assert(VoxelSize() > 0);
  assert(max_tsdf_value > 0);

  Reset();
}

void RegularGridTSDF::Reset() {
  TSDF empty(0, 0, max_tsdf_value_);
  device_grid_.fill(empty);
}

const SimilarityTransform& RegularGridTSDF::GridFromWorld() const {
  return grid_from_world_;
}

const SimilarityTransform& RegularGridTSDF::WorldFromGrid() const {
  return world_from_grid_;
}

Box3f RegularGridTSDF::BoundingBox() const {
  return bounding_box_;
}

Vector3i RegularGridTSDF::Resolution() const {
  return device_grid_.size();
}

float RegularGridTSDF::VoxelSize() const {
  return world_from_grid_.scale;
}

Vector3f RegularGridTSDF::SideLengths() const {
  return VoxelSize() * Resolution();
}

void RegularGridTSDF::Fuse(const Vector4f& depth_camera_flpp,
  const Range1f& depth_range,
  const Matrix4f& camera_from_world,
  DeviceArray2D<float>& depth_data) {

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { device_grid_.width(), device_grid_.height() },
    block_dim
  );

  Event e;
  e.recordStart();
  FuseKernel<<<grid_dim, block_dim>>>(
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    make_float4(depth_camera_flpp),
    make_float2(depth_range.left(), depth_range.right()),
    make_float4x4(camera_from_world),
    depth_data.readView(),
    device_grid_.writeView());
  float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

  printf("Fuse() took : %f ms\n", msElapsed);
}

void RegularGridTSDF::Raycast(const Vector4f& depth_camera_flpp,
  const Matrix4f& world_from_camera,
  DeviceArray2D<float4>& world_points_out,
  DeviceArray2D<float4>& world_normals_out) {

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { world_points_out.width(), world_points_out.height() },
    block_dim
  );

  Vector4f eye = world_from_camera * Vector4f(0, 0, 0, 1);

  Event e;
  e.recordStart();
  RaycastKernel<<<grid_dim, block_dim>>>(
    device_grid_.readView(),
    make_float4x4(grid_from_world_.asMatrix()),
    make_float4x4(world_from_grid_.asMatrix()),
    max_tsdf_value_,
    make_float4(depth_camera_flpp),
    make_float4x4(world_from_camera),
    make_float3(eye.xyz),
    world_points_out.writeView(),
    world_normals_out.writeView()
  );
  float msElapsed = e.recordStopSyncAndGetMillisecondsElapsed();

  printf("Raycast() took: %f ms\n", msElapsed);
}

TriangleMesh RegularGridTSDF::Triangulate() const {
  Array3D<TSDF> host_grid(Resolution());
  device_grid_.copyToHost(host_grid);

  std::vector<Vector3f> positions;
  std::vector<Vector3f> normals;
  MarchingCubes(host_grid, max_tsdf_value_, world_from_grid_,
    positions, normals);

  return ConstructMarchingCubesMesh(positions, normals);
}
