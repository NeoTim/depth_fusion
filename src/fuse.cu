#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "fuse.h"

#include "libcgt/cuda/float4x4.h"
#include "libcgt/cuda/MathUtils.h"
#include "libcgt/cuda/ThreadMath.cuh"

#include "camera_math.cuh"

using libcgt::cuda::threadmath::threadSubscript2DGlobal;
using libcgt::cuda::contains;
using libcgt::cuda::math::roundToInt;

__global__
void FuseKernel(
  float4x4 world_from_grid,
  float max_tsdf_value,
  float4 flpp,
  float2 depth_min_max,
  float4x4 camera_from_world,
  KernelArray2D<const float> depth_map,
  KernelArray3D<TSDF> regular_grid) {

  int2 ij = threadSubscript2DGlobal();

  // Sweep over the entire volume.
  for (int k = 0; k < regular_grid.depth(); ++k) {
    // Find the voxel center.
    // TODO(jiawen): write a helper function that takes in a subscript
    float4 voxel_center_world = make_float4(
      transformPoint(
        world_from_grid, float3{ij.x + 0.5f, ij.y + 0.5f, k + 0.5f}),
      1.0f);

    // Project it into camera coordinates.
    // camera_from_world uses OpenGL conventions,
    // so depth is a negative number if it's in front of the camera.
    float4 voxel_center_camera =
      camera_from_world * voxel_center_world;
    float2 uv = make_float2(PixelFromCamera(make_float3(voxel_center_camera),
      flpp));
    int2 uv_int = roundToInt(uv - float2{ 0.5f, 0.5f });

    if (voxel_center_camera.z > 0 ||
      !contains(depth_map.size(), uv_int)) {
      continue;
    }

    float image_depth = depth_map[uv_int];
    if (image_depth < depth_min_max.x || image_depth > depth_min_max.y) {
      continue;
    }

    // Compute dz, the signed distance between the voxel center and the
    // surface observation.
    //
    // Note that we flip the sign on z to get "depth", where positive numbers
    // are in front of the camera.
    //
    // The sign convention of the distance field is so that voxels in front of
    // the surface is positive (and voxels behind are negative).
    float voxel_center_depth = -voxel_center_camera.z;
    float dz = image_depth - voxel_center_depth;

    // Now integrate data in carefully:
    // Consider 3 cases:
    // dz < -max_tsdf_value: the voxel is behind the observation and out of the
    //   truncation region. Therefore, do nothing.
    // dz \in [-max_tsdf_value, 0]: the voxel is behind the observation and
    //   within the truncation region. Integrate.
    // dz > 0: the voxel is in front of the observation. Integrate... but if
    //   the voxel is really far in front, we don't want to put in a large
    //   value. Instead, clamp it to max_tsdf_value.
    if (dz >= -max_tsdf_value) {
      // Ignore the voxel when it is far behind.
      // Clamp to the TSDF range.
      dz = min(dz, max_tsdf_value);
      const float weight = 1.0f;

      regular_grid[{ij.x, ij.y, k}].Update(dz, weight, max_tsdf_value);
    }
  }
}

namespace {
  constexpr int kNumDepthMaps = 3;
}

__global__
void FuseMultipleKernel(
  float4x4 world_from_grid,
  float max_tsdf_value,
  CalibratedPosedDepthCamera depth_camera0,
  CalibratedPosedDepthCamera depth_camera1,
  CalibratedPosedDepthCamera depth_camera2,
  KernelArray2D<const float> depth_map0,
  KernelArray2D<const float> depth_map1,
  KernelArray2D<const float> depth_map2,
  KernelArray3D<TSDF> regular_grid) {

  CalibratedPosedDepthCamera depth_camera[] =
  {
    depth_camera0,
    depth_camera1,
    depth_camera2
  };
  KernelArray2D<const float> depth_maps[] =
  {
    depth_map0,
    depth_map1,
    depth_map2
  };

  int2 ij = threadSubscript2DGlobal();

  // Sweep over the entire volume.
  for (int k = 0; k < regular_grid.depth(); ++k) {
    // Find the voxel center.
    // TODO(jiawen): write a helper function that takes in a subscript
    float4 voxel_center_world = make_float4(
      transformPoint(
        world_from_grid, float3{ij.x + 0.5f, ij.y + 0.5f, k + 0.5f}),
      1.0f);

    for (int c = 0; c < kNumDepthMaps; ++c) {
      // Project it into camera coordinates.
      // camera_from_world uses OpenGL conventions,
      // so depth is a negative number if it's in front of the camera.
      float4 voxel_center_camera =
        depth_camera[c].camera_from_world * voxel_center_world;
      float2 uv = make_float2(
        PixelFromCamera(make_float3(voxel_center_camera), depth_camera[c].flpp));
      int2 uv_int = roundToInt(uv - float2{0.5f, 0.5f});

      if (voxel_center_camera.z > 0 ||
        !contains( depth_maps[c].size(), uv_int)) {
        continue;
      }

      float image_depth = depth_maps[c][uv_int];
      if (image_depth < depth_camera[c].depth_min_max.x ||
        image_depth > depth_camera[c].depth_min_max.y) {
        continue;
      }

      // Compute dz, the signed distance between the voxel center and the
      // surface observation.
      //
      // Note that we flip the sign on z to get "depth", where positive numbers
      // are in front of the camera.
      //
      // The sign convention of the distance field is so that voxels in front of
      // the surface is positive (and voxels behind are negative).
      float voxel_center_depth = -voxel_center_camera.z;
      float dz = image_depth - voxel_center_depth;

      // Now integrate data in carefully:
      // Consider 3 cases:
      // dz < -max_tsdf_value: the voxel is behind the observation and out of the
      //   truncation region. Therefore, do nothing.
      // dz \in [-max_tsdf_value, 0]: the voxel is behind the observation and
      //   within the truncation region. Integrate.
      // dz > 0: the voxel is in front of the observation. Integrate... but if
      //   the voxel is really far in front, we don't want to put in a large
      //   value. Instead, clamp it to max_tsdf_value.
      if (dz >= -max_tsdf_value) {
        // Ignore the voxel when it is far behind.
        // Clamp to the TSDF range.
        dz = min(dz, max_tsdf_value);
        const float weight = 1.0f;

        regular_grid[{ij.x, ij.y, k}].Update(dz, weight, max_tsdf_value);
      }
    }
  }
}
