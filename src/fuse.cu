#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "fuse.h"

#include <cuda/float4x4.h>
#include <cuda/MathUtils.h>
#include <cuda/ThreadMath.cuh>

#include "camera_math.cuh"

using libcgt::cuda::threadmath::threadSubscript2DGlobal;
using libcgt::cuda::contains;
using libcgt::cuda::math::roundToInt;

#define USE_BILINEAR_DEPTH_SAMPLING 1

// TODO(jiawen): move this into class regular_grid_tsdf.
__global__
void FuseKernel(
  float4x4 world_from_grid,
  float max_tsdf_value,
  float4 flpp,
  float2 zMinMax,
  float4x4 camera_from_world,
  KernelArray2D<const float> depth_data,
  KernelArray3D<TSDF> regular_grid) {

  int2 ij = threadSubscript2DGlobal();

  // Sweep over the entire volume.
  for (int k = 0; k < regular_grid.depth(); ++k) {
    // Find the voxel center.
    // TODO(jiawen): write a helper function that takes in a subscript
    float4 voxel_center_world = make_float4(
      transformPoint(
        world_from_grid, float3{ij.x + 0.5f, ij.y + 0.5f, k + 0.5f}),
      1.0f);

    // Project it into camera coordinates.
    // camera_from_world uses OpenGL conventions,
    // so depth is a negative number if it's in front of the camera.
    float4 voxel_center_camera =
      camera_from_world * voxel_center_world;
    float2 uv = make_float2(PixelFromCamera(make_float3(voxel_center_camera),
      flpp));

#if USE_BILINEAR_DEPTH_SAMPLING
    // Use bilinear interpolation.
    int2 uv_int = roundToInt(uv);

    if (voxel_center_camera.z > 0 ||
        uv_int.x < 0 || uv_int.y < 0 ||
	uv_int.x + 1 >= depth_data.size().x || uv_int.y + 1 >= depth_data.size().y) {
      continue;
    }

    float2 fuv = uv - float2{uv_int.x, uv_int.y};
    float d00 = depth_data[uv_int + int2{0, 0}];
    float d10 = depth_data[uv_int + int2{1, 0}];
    float d01 = depth_data[uv_int + int2{0, 1}];
    float d11 = depth_data[uv_int + int2{1, 1}];

    // Check if all depth measurements are within the range.
    if (min(min(d00,d01),min(d10,d11)) < zMinMax.x ||
	max(max(d00,d01),max(d10,d11)) > zMinMax.y) {
      continue;
    }

    float image_depth = (1 - fuv.y) * (d00 * (1 - fuv.x) + fuv.x * d10) +
	                fuv.y * (d01 * (1 - fuv.x) + fuv.x * d11);
#else
    // Use nearest sampling on the depth texture.
    int2 uv_int = roundToInt(uv - float2{ 0.5f, 0.5f });

    if (voxel_center_camera.z > 0 ||
      !contains(depth_data.size(), uv_int)) {
      continue;
    }

    float image_depth = depth_data[uv_int];
#endif

    if (image_depth < zMinMax.x || image_depth > zMinMax.y) {
      continue;
    }

    // Compute standard deviation of depth in a squared patch around the sample.
    // This value is can be used a proxy for the weight of each sample,
    // so the samples around depth discontinuties or grazing angles can be
    // downsampled to avoid exaggerated space carving artifacts.
    const int kDepthVarRadius = 3;

    if (uv_int.x - kDepthVarRadius < 0 || uv_int.y - kDepthVarRadius < 0 ||
	uv_int.x + kDepthVarRadius >= depth_data.size().x ||
	uv_int.y + kDepthVarRadius  >=depth_data.size().y) {
      continue;
    }

    float sum_d = 0.0f, sum_d2 = 0.0f;
    for (int dx = -kDepthVarRadius; dx <= kDepthVarRadius; dx++) {
      for (int dy = -kDepthVarRadius; dy <= kDepthVarRadius; dy++) {
        float d = depth_data[uv_int + int2{dx, dy}];
        sum_d += d;
        sum_d2 += d * d;
      }
    }
    const int var_n = (2 * kDepthVarRadius + 1) * (2 * kDepthVarRadius + 1);
    float var_d = sum_d2 / var_n - sum_d * sum_d / var_n / var_n;
    float std_d = sqrt(var_d);

    // Compute dz, the signed distance between the voxel center and the
    // surface observation.
    //
    // Note that we flip the sign on z to get "depth", where positive numbers
    // are in front of the camera.
    //
    // The sign convention of the distance field is so that voxels in front of
    // the surface is positive (and voxels behind are negative).
    float voxel_center_depth = -voxel_center_camera.z;
    float dz = image_depth - voxel_center_depth;

    // Now integrate data in carefully:
    // Consider 3 cases:
    // dz < -max_tsdf_value: the voxel is behind the observation and out of the
    //   truncation region. Therefore, do nothing.
    // dz \in [-max_tsdf_value, 0]: the voxel is behind the observation and
    //   within the truncation region. Integrate.
    // dz > 0: the voxel is in front of the observation. Integrate... but if
    //   the voxel is really far in front, we don't want to put in a large
    //   value. Instead, clamp it to max_tsdf_value.
    if (dz >= -max_tsdf_value) {
      // Ignore the voxel when it is far behind.
      // Clamp to the TSDF range.
      dz = min(dz, max_tsdf_value);

      // Weights can be larger than 1.0, the absolute value only matters in
      // the marching cubes algorithm, where a threshold on the weight decides
      // where to create a triangle.
      // The weight is higher the less standard deviation in a square patch
      // around surface.
      float weight = 10.0f * min(0.1f / std_d, 1.0f);

      regular_grid[{ij.x, ij.y, k}].Update(dz, weight, max_tsdf_value);
    }
  }
}
