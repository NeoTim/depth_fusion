#include "hip/hip_runtime.h"
// Copyright 2016 Google Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http ://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "projective_point_plane_icp.h"

#include <chrono>

#include <hip/hip_vector_types.h>

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include <core/vecmath/Quat4f.h>
#include <core/time/TimeUtils.h>
#include <cuda/Event.h>
#include <cuda/MathUtils.h>
#include <cuda/Rect2i.h>
#include <cuda/ThreadMath.cuh>
#include <cuda/VecmathConversions.h>

#include "camera_math.cuh"

using libcgt::cuda::contains;
using libcgt::cuda::math::floorToInt;
using libcgt::cuda::Rect2i;
using libcgt::cuda::threadmath::threadSubscript2DGlobal;
using libcgt::core::vecmath::EuclideanTransform;

// TODO(jiawen): can potentially optimize this.
struct Plus {
  __host__ __device__
    ICPLeastSquaresData operator () (const ICPLeastSquaresData& lhs,
      const ICPLeastSquaresData& rhs) const {
    ICPLeastSquaresData sum;
    for (int i = 0; i < 21; ++i) {
      sum.a[i] = lhs.a[i] + rhs.a[i];
    }
    for (int i = 0; i < 6; ++i) {
      sum.b[i] = lhs.b[i] + rhs.b[i];
    }
    sum.num_samples = lhs.num_samples + rhs.num_samples;
    sum.squared_residual = lhs.squared_residual + rhs.squared_residual;
    return sum;
  }
};

// TODO(jiawen): make a version without debug output
__global__
void ICPKernel(
  float4 flpp, // depth camera intrinsics
  float2 depth_min_max,
  float4x4 model_from_world,   // known model pose
  float4x4 model_from_current, // current pose estimate
  float4x4 current_from_model, // current pose estimate
  KernelArray2D<const float> depth_map,
  KernelArray2D<const float4> normal_map,
  KernelArray2D<const float4> world_points,
  KernelArray2D<const float4> world_normals,
  int src_image_guard_band_pixels,
  float max_distance_for_match,
  float min_dot_product_for_match,
  KernelArray2D<ICPLeastSquaresData> icp_data_out,
  KernelArray2D<uchar4> debug_vis_out) {
  // TODO: weighting function parameters
  // Reject data association if position differs by more than eps1
  // and normal dot product more than eps2?

  int2 dst_xy = threadSubscript2DGlobal();
  ICPLeastSquaresData output = {};
  uchar4 debug_output = {};

  float4 dst_point_world4 = world_points[dst_xy];
  float4 dst_normal_world4 = world_normals[dst_xy];

  if (dst_point_world4.w == 0 || dst_normal_world4.w == 0) {
    icp_data_out[dst_xy] = output;
    debug_vis_out[dst_xy] = debug_output;
    return;
  }

  float3 dst_point_world = make_float3(dst_point_world4);
  float3 dst_normal_world = make_float3(dst_normal_world4);

  float3 dst_point_model = transformPoint(model_from_world, dst_point_world);
  float3 dst_normal_model = transformVector(model_from_world, dst_normal_world);

  // Project dst_point into current pose estimate to see if it associates.
  float3 dst_point_current = transformPoint(current_from_model, dst_point_model);
  int2 dst_xy_current = floorToInt(make_float2(PixelFromCamera(
    dst_point_current, flpp)));

  Rect2i valid_rect = inset(Rect2i(depth_map.size()),
    src_image_guard_band_pixels);
  // If the point is in front of the camera, then dst_point_current.z < 0.
  if (!contains(valid_rect, dst_xy_current) || dst_point_current.z > 0) {
    icp_data_out[dst_xy] = output;
    debug_vis_out[dst_xy] = uchar4{ 255, 0, 0, 255 };
    return;
  }

  float src_depth = depth_map[dst_xy_current];
  float4 src_normal_current4 = normal_map[dst_xy_current];

  if (src_depth < depth_min_max.x || src_depth > depth_min_max.y ||
    src_normal_current4.w == 0) {
    icp_data_out[dst_xy] = output;
    debug_vis_out[dst_xy] = uchar4{ 0, 255, 0, 255 };
    return;
  }

  // Unproject src pixel into camera coordinates and then into model camera
  // coordinates.
  float3 src_point_current = CameraFromPixel(dst_xy_current, src_depth, flpp);
  float3 src_point_model = transformPoint(model_from_current,
    src_point_current);
  float3 src_normal_model = transformVector(model_from_current,
    make_float3(src_normal_current4));

  // TODO(jiawen): write a parameterized weight function which accepts points,
  // including 0.
  float3 delta = dst_point_model - src_point_model;
  if (length(delta) > max_distance_for_match) {
    icp_data_out[dst_xy] = output;
    debug_vis_out[dst_xy] = uchar4{ 0, 0, 255, 255 };
    return;
  }

  if (dot(src_normal_model, dst_normal_model) < min_dot_product_for_match) {
    icp_data_out[dst_xy] = output;
    debug_vis_out[dst_xy] = uchar4{ 255, 255, 0, 255 };
    return;
  }

  // Declare sample as valid.
  output.num_samples = 1;

  float3 c = cross(src_point_model, dst_normal_model);
  float r = dot(delta, dst_normal_model);

  output.a[ 0] = c.x * c.x;
  output.a[ 1] = c.y * c.x;
  output.a[ 2] = c.z * c.x;
  output.a[ 3] = dst_normal_model.x * c.x;
  output.a[ 4] = dst_normal_model.y * c.x;
  output.a[ 5] = dst_normal_model.z * c.x;

  output.a[ 6] = c.y * c.y;
  output.a[ 7] = c.z * c.y;
  output.a[ 8] = dst_normal_model.x * c.y;
  output.a[ 9] = dst_normal_model.y * c.y;
  output.a[10] = dst_normal_model.z * c.y;

  output.a[11] = c.z * c.z;
  output.a[12] = dst_normal_model.x * c.z;
  output.a[13] = dst_normal_model.y * c.z;
  output.a[14] = dst_normal_model.z * c.z;

  output.a[15] = dst_normal_model.x * dst_normal_model.x;
  output.a[16] = dst_normal_model.y * dst_normal_model.x;
  output.a[17] = dst_normal_model.z * dst_normal_model.x;

  output.a[18] = dst_normal_model.y * dst_normal_model.y;
  output.a[19] = dst_normal_model.z * dst_normal_model.y;

  output.a[20] = dst_normal_model.z * dst_normal_model.z;

  output.b[0] = c.x * r;
  output.b[1] = c.y * r;
  output.b[2] = c.z * r;
  output.b[3] = dst_normal_model.x * r;
  output.b[4] = dst_normal_model.y * r;
  output.b[5] = dst_normal_model.z * r;

  output.squared_residual = r * r;

  icp_data_out[dst_xy] = output;
  debug_vis_out[dst_xy] = uchar4{ 255, 255, 255, 255 };
}

Matrix4f rigidTransformationFromApprox(float x[6]) {
  float alpha = x[0];
  float beta  = x[1];
  float gamma = x[2];

  // TODO(jiawen): investigate using this one instead, it should work too and
  // is more symmetric.
#if 0
  return Matrix4f
  (
    1.0f,   -gamma,   beta,   x[3],
    gamma,    1.0f, -alpha,   x[4],
    -beta,   alpha,   1.0f,   x[5],
    0.0f,     0.0f,   0.0f,   1.0f
  );
#else
  return Matrix4f::translation({ x[3], x[4], x[5] }) *
    Matrix4f::rotateZ(gamma) *
    Matrix4f::rotateY(beta) *
    Matrix4f::rotateX(alpha);
#endif
}

ProjectivePointPlaneICP::ProjectivePointPlaneICP(
  const Vector2i& depth_resolution,
  const Intrinsics& depth_intrinsics, const Range1f& depth_range) :
  icp_data_(depth_resolution),
  depth_intrinsics_flpp_{ depth_intrinsics.focalLength,
    depth_intrinsics.principalPoint },
  depth_range_(depth_range) {
}

// TODO(jiawen): can improve conditioning by subtracting off the mean first
// This would make c = p x n smaller.

// Estimate camera pose using ICP.
__host__
ProjectivePointPlaneICP::Result ProjectivePointPlaneICP::EstimatePose(
  DeviceArray2D<float>& incoming_depth,
  DeviceArray2D<float4>& incoming_normals,
  const EuclideanTransform& world_from_camera,
  DeviceArray2D<float4>& world_points,
  DeviceArray2D<float4>& world_normals,
  DeviceArray2D<uchar4>& debug_vis) {
  auto t0 = std::chrono::high_resolution_clock::now();

  dim3 block_dim(16, 16, 1);
  dim3 grid_dim = libcgt::cuda::math::numBins2D(
    { incoming_depth.width(), incoming_depth.height() },
    block_dim
  );

  const ICPLeastSquaresData zero = {};

  ProjectivePointPlaneICP::Result result;

  const float4x4 model_from_world = make_float4x4(
    inverse(world_from_camera).asMatrix());
  Matrix4f model_from_current = Matrix4f::identity();
  for (int i = 0; i < kNumIterations; ++i) {
    Matrix4f current_from_model =
      Matrix4f::inverseEuclidean(model_from_current);

    ICPKernel<<<grid_dim, block_dim>>>(
      make_float4(depth_intrinsics_flpp_),
      make_float2(depth_range_.leftRight()),
      model_from_world,
      make_float4x4(model_from_current),
      make_float4x4(current_from_model),
      incoming_depth.readView(),
      incoming_normals.readView(),
      world_points.readView(),
      world_normals.readView(),
      kImageGuardBand,
      kMaxDistanceForMatch,
      kMinDotProductForMatch,
      icp_data_.writeView(),
      debug_vis.writeView());

    ICPLeastSquaresData* begin = icp_data_.pointer();
    ICPLeastSquaresData* end = icp_data_.rowPointer(icp_data_.height());
    // TODO(jiawen): benchmark how long the sum takes. Look into if the sum
    // can be computed on the GPU as well as the inversion.
    ICPLeastSquaresData sum =
      thrust::reduce(thrust::device, begin, end, zero, Plus());

    printf("ICP iteration %d (after assoc, before solve)\n"
      "num_samples = %d, squared_residual = %f\n",
      i, sum.num_samples, sum.squared_residual);

    if (sum.num_samples < kMinNumSamples) {
      result.valid = false;
      result.num_samples = sum.num_samples;
      return result;
    }

    // TODO(jiawen): benchmark how long the solve takes.
    float x[6];
    Solve(sum, x);
    Matrix4f incremental = rigidTransformationFromApprox(x);
    model_from_current = incremental * model_from_current;
  }

  Quat4f q = Quat4f::fromRotationMatrix(model_from_current.getSubmatrix3x3());
  Vector3f t = model_from_current.getCol(3).xyz;
  float radians;
  Vector3f axis = q.getAxisAngle(&radians);

  if (t.norm() > kMaxTranslation || radians > kMaxRotationRadians) {
    result.valid = false;
    return result;
  }

  result.valid = true;

  Matrix4f new_world_from_camera = world_from_camera.asMatrix() * model_from_current;
  Vector4f eye = new_world_from_camera * Vector4f(0, 0, 0, 1);
  Vector4f center = new_world_from_camera * Vector4f(0, 0, -1, 1);
  Vector4f up = new_world_from_camera * Vector4f(0, 1, 0, 0);

  Matrix4f camera_from_world = Matrix4f::lookAt(eye.xyz, center.xyz, up.xyz);
  result.world_from_camera = EuclideanTransform::fromMatrix(
    Matrix4f::inverseEuclidean(camera_from_world));

  auto t1 = std::chrono::high_resolution_clock::now();
  printf("ICP took %lld ms\n", libcgt::core::time::dtMS(t0, t1));

  return result;
}
